#include "hip/hip_runtime.h"
/* Author: Lingni Ma
 * Created Date: 12 Jan 2016
 * Last Edit: 12 Jan 2016
 * Function: memorized unpooling
 * Note: implementation is based on the UpsampleLayer from SegNet (https://github.com/alexgkendall/caffe-segnet)
 */

#include <algorithm>
#include <cfloat>
#include <vector>
#include <iostream>

#include "caffe/layers/unpooling_layer.hpp"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
  __global__ void unpool_forward_kernel(const int nthreads, int in_w, int in_h,
      int out_w, int out_h, const Dtype* bottom_data,
      const Dtype* bottom_mask, Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      int offset = index / (in_w * in_h) * out_w * out_h;
      int unpool_idx = static_cast<int>(bottom_mask[index]);
      top_data[offset + unpool_idx] = bottom_data[index];
    }
  }

template <typename Dtype>
void UnpoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[DataBlobIndex]->gpu_data();
  const Dtype* bottom_mask = bottom[MaskBlobIndex]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_set(top[0]->count(), Dtype(0), top_data);
  int bottom_count = bottom[DataBlobIndex]->count();
  unpool_forward_kernel<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
      bottom_count, bottom[DataBlobIndex]->width(), bottom[DataBlobIndex]->height(),
      top[0]->width(), top[0]->height(), bottom_data, bottom_mask, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
  __global__ void unpool_backward_kernel(const int nthreads, int in_w, int in_h,
      int out_w, int out_h, const Dtype* top_diff,
      const Dtype* bottom_mask, Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      int offset = index / (in_w * in_h) * out_w * out_h;
      int unpool_idx = static_cast<int>(bottom_mask[index]);
      bottom_diff[index] = top_diff[offset + unpool_idx];
    }
  }

template <typename Dtype>
void UnpoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_mask = bottom[MaskBlobIndex]->gpu_data();
    Dtype* bottom_diff = bottom[DataBlobIndex]->mutable_gpu_diff();
    const int bottom_count = bottom[DataBlobIndex]->count();
    caffe_gpu_set(bottom_count, Dtype(0.), bottom_diff);
    unpool_backward_kernel<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom_count, bottom[DataBlobIndex]->width(), bottom[DataBlobIndex]->height(),
        top[0]->width(), top[0]->height(), top_diff, bottom_mask, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(UnpoolingLayer);


}  // namespace caffe
